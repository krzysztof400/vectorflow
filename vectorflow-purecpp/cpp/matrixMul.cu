#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "matrix_utils.h"

__global__ void matrixMultiplyKernel(const float* input, const float* weights, const float* biases, float* output, int inputSize, int outputSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < outputSize) {
        float z_val = biases[idx];
        for (int k = 0; k < inputSize; k++) {
            z_val += input[k] * weights[k * outputSize + idx];
        }
        output[idx] = z_val;
    }
}

void matrixMultiplyCUDA(const float* input, const float* weights, const float* biases, float* output, int inputSize, int outputSize) {
    // Allocate device memory
    float *d_input, *d_weights, *d_biases, *d_output;
    hipMalloc(&d_input, inputSize * sizeof(float));
    hipMalloc(&d_weights, inputSize * outputSize * sizeof(float));
    hipMalloc(&d_biases, outputSize * sizeof(float));
    hipMalloc(&d_output, outputSize * sizeof(float));

    // Copy data to device
    hipMemcpy(d_input, input, inputSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, inputSize * outputSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_biases, biases, outputSize * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int gridSize = (outputSize + blockSize - 1) / blockSize;
    matrixMultiplyKernel<<<gridSize, blockSize>>>(d_input, d_weights, d_biases, d_output, inputSize, outputSize);

    // Copy result back to host
    hipMemcpy(output, d_output, outputSize * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_biases);
    hipFree(d_output);
}